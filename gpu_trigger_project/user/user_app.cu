// gpu_trigger_user.cu
// Build: nvcc -O2 -o gpu_trigger_user gpu_trigger_user.cu
// Run: sudo ./gpu_trigger_user <mode>
// mode: 0=IRQ, 1=IOCTL, 2=Sysfs-direct

#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/ioctl.h>
#include <string.h>
#include <errno.h>
#include <time.h>

#include <hip/hip_runtime.h>

#define DEVICE_PATH      "/dev/gpio_trigger"
#define SYSFS_MODE_DIR   "/sys/class/gpio_trigger_class_unique"
#define SYSFS_MODE_FILE  SYSFS_MODE_DIR "/mode"
#define GPU_TRIGGER_IOCTL _IO('K', 1)

static void checkCuda(hipError_t err, const char* where) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error at %s: %s\n", where, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

static void print_cuda_device_info() {
    int devCount = 0;
    hipError_t err = hipGetDeviceCount(&devCount);
    if (err != hipSuccess) {
        fprintf(stderr, "hipGetDeviceCount error: %s\n", hipGetErrorString(err));
        return;
    }
    printf("[CUDA] Device count: %d\n", devCount);
    for (int d = 0; d < devCount; ++d) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, d);
        printf("[CUDA] Device %d: %s (SMs=%d, totalMem=%.2f MB)\n",
               d, prop.name, prop.multiProcessorCount, prop.totalGlobalMem / (1024.0*1024.0));
    }
}

// Different kernels / sizes for modes
__global__ void vec_add_kernel(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}

void run_cuda_vec_add(int N) {
    size_t bytes = (size_t)N * sizeof(float);
    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);
    if (!h_A || !h_B || !h_C) {
        fprintf(stderr, "Host malloc failed\n");
        free(h_A); free(h_B); free(h_C);
        return;
    }

    // init inputs so result won't be all zeros
    for (int i = 0; i < N; ++i) {
        h_A[i] = (float)i * 1.0f;
        h_B[i] = (float)i * 2.0f;
    }

    float *d_A = NULL, *d_B = NULL, *d_C = NULL;
    checkCuda(hipMalloc((void**)&d_A, bytes), "hipMalloc d_A");
    checkCuda(hipMalloc((void**)&d_B, bytes), "hipMalloc d_B");
    checkCuda(hipMalloc((void**)&d_C, bytes), "hipMalloc d_C");

    checkCuda(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice), "hipMemcpy H2D d_A");
    checkCuda(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice), "hipMemcpy H2D d_B");

    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    vec_add_kernel<<<blocks, threads>>>(d_A, d_B, d_C, N);

    // synchronize & check kernel error
    checkCuda(hipDeviceSynchronize(), "hipDeviceSynchronize after kernel");
    hipError_t kerr = hipGetLastError();
    if (kerr != hipSuccess) {
        fprintf(stderr, "Kernel launch error: %s\n", hipGetErrorString(kerr));
    }

    checkCuda(hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost), "hipMemcpy D2H d_C");

    // print a few samples
    printf("[CUDA] N=%d sample: C[0]=%f, C[%d]=%f\n", N, h_C[0], N-1, h_C[N-1]);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);
}

int write_sysfs_mode(int mode) {
    int fd = open(SYSFS_MODE_FILE, O_WRONLY);
    if (fd < 0) {
        fprintf(stderr, "Failed to open sysfs mode '%s': %s\n", SYSFS_MODE_FILE, strerror(errno));
        return -1;
    }
    char buf[16];
    int len = snprintf(buf, sizeof(buf), "%d\n", mode);
    if (write(fd, buf, len) != len) {
        fprintf(stderr, "Failed to write mode to sysfs: %s\n", strerror(errno));
        close(fd);
        return -1;
    }
    close(fd);
    return 0;
}

ssize_t read_device_once(int devfd, char *buf, size_t bufsize) {
    // read from device: driver returns result_size bytes if available
    lseek(devfd, 0, SEEK_SET); // reset file pos
    ssize_t r = read(devfd, buf, bufsize-1);
    if (r > 0) buf[r] = '\0';
    return r;
}

int main(int argc, char **argv) {
    if (argc < 2) {
        fprintf(stderr, "Usage: %s <mode>\n 0=IRQ 1=IOCTL 2=Sysfs\n", argv[0]);
        return 1;
    }

    int mode = atoi(argv[1]);
    if (mode < 0 || mode > 2) {
        fprintf(stderr, "Invalid mode %d\n", mode);
        return 1;
    }

    print_cuda_device_info();
    // select device 0 by default (adjust if you want another)
    hipError_t err = hipSetDevice(0);
    if (err != hipSuccess) {
        fprintf(stderr, "cudaSetDevice(0) failed: %s\n", hipGetErrorString(err));
        // still continue; cudaMalloc will fail if invalid
    }

    // open char device
    int fd = open(DEVICE_PATH, O_RDWR | O_SYNC);
    if (fd < 0) {
        fprintf(stderr, "Failed to open device '%s': %s\n", DEVICE_PATH, strerror(errno));
        return 1;
    }

    printf("gpu_trigger_user: running for mode %d\n", mode);

    char devbuf[256];
    ssize_t r;

    if (mode == 1) {
        // IOCTL mode: set sysfs mode then issue ioctl once and read result
        if (write_sysfs_mode(1) != 0) { close(fd); return 1; }
        printf("[Mode 1] writing sysfs mode=1 and issuing IOCTL\n");
        if (ioctl(fd, GPU_TRIGGER_IOCTL) < 0) {
            fprintf(stderr, "ioctl failed: %s\n", strerror(errno));
        } else {
            // wait small time for kernel work to finish; driver writes result quickly
            for (int i = 0; i < 10; ++i) {
                r = read_device_once(fd, devbuf, sizeof(devbuf));
                if (r > 0) break;
                usleep(100000); // 100ms
            }
            if (r > 0) {
                printf("[Driver result] %s", devbuf);
            } else {
                printf("[Driver result] no result available after ioctl\n");
            }
            // Launch CUDA with mode-dependent size
            int N = 1<<20; // medium
            run_cuda_vec_add(N);
        }
    }
    else if (mode == 2) {
        // Sysfs trigger: write mode=2, kernel triggers immediately; read result then run CUDA
        if (write_sysfs_mode(2) != 0) { close(fd); return 1; }
        printf("[Mode 2] wrote sysfs mode=2 (kernel triggers work)\n");
        // wait for device result
        for (int i = 0; i < 20; ++i) {
            r = read_device_once(fd, devbuf, sizeof(devbuf));
            if (r > 0) break;
            usleep(100000);
        }
        if (r > 0) {
            printf("[Driver result] %s", devbuf);
        } else {
            printf("[Driver result] no result available after sysfs write\n");
        }
        // Mode 2: bigger job
        int N = 1<<22; // ~4M elements (be careful with memory)
        // reduce if your GPU memory is small:
        size_t approx_bytes = (size_t)N * sizeof(float);
        if (approx_bytes > (size_t)1024ull*1024ull*1024ull) {
            // if >1GB, reduce
            N = 1<<20;
            printf("[Mode2] large job reduced to N=%d to fit memory\n", N);
        }
        run_cuda_vec_add(N);
    }
    else { // mode == 0 (IRQ)
        if (write_sysfs_mode(0) != 0) { close(fd); return 1; }
        printf("[Mode 0] waiting for kernel to produce result (driver's IRQ handling)...\n");
        // Poll/read loop: only run CUDA when driver read returns >0
        while (1) {
            r = read_device_once(fd, devbuf, sizeof(devbuf));
            if (r > 0) {
                printf("[Driver result] %s", devbuf);
                // Small job for interrupt mode
                int N = 1<<16;
                run_cuda_vec_add(N);
            }
            // sleep a bit to avoid busy loop
            usleep(200000); // 200ms
        }
    }

    close(fd);
    return 0;
}